#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cudaLinearAlgebra.hpp>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}


__global__ void k_dot_product(int *res, int *m1, int *m2, int nrow1, int ncol1, int nrow2, int ncol2) {

  int			id = blockDim.x * blockIdx.x + threadIdx.x;

  if (id < (nrow1 * ncol2 * ncol1)) {
    // product
    res_row_idx = (id / ncol1) / ncol2;
    res_col_idx = (id / ncol1) % ncol2;
    idx_in_cell = (id) % ncol1;
    res[id] = m1[res_row_idx][idx_in_cell] * m2[idx_in_cell][res_col_idx];

    __syncthreads();
    // sum reduce
    for (int i = 2; i < ncol2; i * 2) {
      
    }
  }
}


int           *dot_product(int *m1, int *m2, int nrow1, int ncol1, int nrow2, int ncol2) {

  int         *h_res = (int *)malloc(sizeof(int) * (nrow1 * ncol2));
  int         *d_m1;
  int         **d_m1_ = &d_m1;
  int         *d_m2;
  int         **d_m2_ = &d_m2;
  int         *d_res;
  int         **d_res_ = &d_res;

  int		blocks = ((nrow1 * ncol2) % NB_THREADS == 0) ? ((nrow1 * ncol2 * ncol1) / NB_THREADS):((nrow1 * ncol2) / NB_THREADS) + 1;

  checkCudaErrors(hipMalloc(d_m1_, sizeof(int) * len));
  checkCudaErrors(hipMalloc(d_m2_, sizeof(int) * len));
  checkCudaErrors(hipMalloc(d_res_, sizeof(int) * len));
  checkCudaErrors(hipMemcpy(d_m1, v, sizeof(int) * (nrow1 * ncol1), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_m2, v, sizeof(int) * (nrow2 * ncol2), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemset(d_res, 0, sizeof(int) * (nrow1 * ncol2 * ncol1)));

  k_dot_product<<<blocks, NB_THREADS>>>(d_res, d_m1, d_m2, nrow1, ncol1, nrow2, ncol2);

  hipDeviceSynchronize();// checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipMemcpy(h_result, d_result, sizeof(int) * len, hipMemcpyDeviceToHost));

  hipFree(d_m1_);
  hipFree(d_m2_);
  hipFree(d_res_);

  return (h_res);
}
