#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cudaLinearAlgebra.hpp>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}


__global__ void k_dot_product(int *res, int *m1, int *m2, int nrow1, int ncol1, int nrow2, int ncol2) {

  int			id = blockDim.x * blockIdx.x + threadIdx.x;

  if (id < (nrow1 * ncol2 * ncol1)) {
    // product
    int res_row_idx = (id / ncol1) / ncol2;
    int res_col_idx = (id / ncol1) % ncol2;
    int idx_in_cell = (id) % ncol1;

    res[id] = m1[res_row_idx * ncol1 + idx_in_cell] * m2[idx_in_cell * ncol2 + res_col_idx];

    __syncthreads();

    // sum reduce
    int s = ncol1;
    for (int i = ncol1 / 2; i > 0; i >>= 1) {
      if (idx_in_cell < i) {
        // printf("idx_in_cell : %d -- res[%d] += res[%d + %d] -> %d += %d\n", idx_in_cell, id, id, i, res[id], res[id + i]);
        res[id] += res[id + i];
        if (s % 2 == 1 && idx_in_cell == (i - 1))
          res[id] += res[id + i + 1];
      }
      else
        break ;
      s = i;
      __syncthreads();
    }
  }
}

  // 1  2  3    5  2  10 1      a  c  e  g
  // 4  5  6    6  12 8  3      b  d  f  h
  //            9  4  11 6

  // [5][12][27] [2][24][12] [10][16][33] ...
  // [44]        [38]        [59]         ...


int           *dot_product(int *m1, int *m2, int nrow1, int ncol1, int nrow2, int ncol2) {

  int         *h_res = (int *)malloc(sizeof(int) * (nrow1 * ncol2) * ncol1);
  int         *res = (int *)malloc(sizeof(int) * (nrow1 * ncol2));
  int         *d_m1;
  int         **d_m1_ = &d_m1;
  int         *d_m2;
  int         **d_m2_ = &d_m2;
  int         *d_res;
  int         **d_res_ = &d_res;

  int		blocks = ((nrow1 * ncol2) % NB_THREADS == 0) ? ((nrow1 * ncol2 * ncol1) / NB_THREADS):((nrow1 * ncol2) / NB_THREADS) + 1;

  checkCudaErrors(hipMalloc(d_m1_, sizeof(int) * (nrow1 * ncol1)));
  checkCudaErrors(hipMalloc(d_m2_, sizeof(int) * (nrow2 * ncol2)));
  checkCudaErrors(hipMalloc(d_res_, sizeof(int) * (nrow1 * ncol2 * ncol1)));
  checkCudaErrors(hipMemcpy(d_m1, m1, sizeof(int) * (nrow1 * ncol1), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_m2, m2, sizeof(int) * (nrow2 * ncol2), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemset(d_res, 0, sizeof(int) * (nrow1 * ncol2 * ncol1)));

  k_dot_product<<<blocks, NB_THREADS>>>(d_res, d_m1, d_m2, nrow1, ncol1, nrow2, ncol2);

  hipDeviceSynchronize();// checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipMemcpy(h_res, d_res, sizeof(int) * (nrow1 * ncol2 * ncol1), hipMemcpyDeviceToHost));

  hipFree(d_m1_);
  hipFree(d_m2_);
  hipFree(d_res_);

  for (int i = 0; i < nrow1 * ncol2; i++) {
    res[i] = h_res[i * ncol1];
  }
  return (res);
}
