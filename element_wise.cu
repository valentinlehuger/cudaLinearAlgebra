#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define NB_THREADS 50

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

enum ft_op {ADD = 0, SUB, MUL, DIV, MOD};



template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}


__global__ void	k_addition_element_wise(int *vec, int len, int n) {

	int			id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < len)
		vec[id] += n;
}

__global__ void	k_addition_element_wise(int *result, int *vec1, int *vec2, int len) {

	int			id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < len)
		result[id] = vec1[id] + vec2[id];
}

__global__ void	k_substraction_element_wise(int *vec, int len, int n) {

	int			id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < len)
		vec[id] -= n;
}

__global__ void	k_multiplication_element_wise(int *vec, int len, int n) {

	int			id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < len)
		vec[id] *= n;
}

__global__ void	k_division_element_wise(int *vec, int len, int n) {

	int			id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < len)
		vec[id] /= n;
}

__global__ void	k_modulo_element_wise(int *vec, int len, int n) {

	int			id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < len)
		vec[id] = vec[id] % n;
}

void			element_wise(int *v, int len, int n, ft_op op) {

	int		*d_vec;
	int		**d_vec_ = &d_vec;

	int		blocks = (len % NB_THREADS == 0) ? (len / NB_THREADS):(len / NB_THREADS) + 1;

	checkCudaErrors(hipMalloc(d_vec_, sizeof(int) * len));
	checkCudaErrors(hipMemcpy(d_vec, v, sizeof(int) * len, hipMemcpyHostToDevice));

	if (op == ADD)
		k_addition_element_wise<<<blocks, NB_THREADS>>>(d_vec, len, n);
	else if (op == SUB)
		k_substraction_element_wise<<<blocks, NB_THREADS>>>(d_vec, len, n);
	else if (op == MUL)
		k_multiplication_element_wise<<<blocks, NB_THREADS>>>(d_vec, len, n);
	else if (op == DIV)
		k_division_element_wise<<<blocks, NB_THREADS>>>(d_vec, len, n);
	else if (op == MOD)
		k_modulo_element_wise<<<blocks, NB_THREADS>>>(d_vec, len, n);

	hipDeviceSynchronize();// checkCudaErrors(cudaGetLastError());
	checkCudaErrors(hipMemcpy(v, d_vec, sizeof(int) * len, hipMemcpyDeviceToHost));
	hipFree(d_vec_);
}

/*
**	Element wise function for vector vector operation
**	Returns a int array containing the operation result
*/

int			*element_wise(int *v1, int *v2, int len, ft_op op) {

	int		*d_vec1;
	int		*d_vec2;
	int		**d_vec1_ = &d_vec1;
	int		**d_vec2_ = &d_vec2;
	int		*h_result = (int *)malloc(sizeof(int) * len);
	int		*d_result;
	int		**d_result_ = &d_result;


	int		blocks = (len % NB_THREADS == 0) ? (len / NB_THREADS):(len / NB_THREADS) + 1;

	checkCudaErrors(hipMalloc(d_vec1_, sizeof(int) * len));
	checkCudaErrors(hipMalloc(d_vec2_, sizeof(int) * len));
	checkCudaErrors(hipMalloc(d_result_, sizeof(int) * len));
	checkCudaErrors(hipMemcpy(d_vec1, v1, sizeof(int) * len, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_vec2, v2, sizeof(int) * len, hipMemcpyHostToDevice));
	// checkCudaErrors(cudaMemcpy(d_result, h_result, sizeof(int) * len, cudaMemcpyHostToDevice));
	checkCudaErrors(hipMemset(d_result, 0, sizeof(int) * len));

	if (op == ADD)
		k_addition_element_wise<<<blocks, NB_THREADS>>>(d_result, d_vec1, d_vec2, len);
	// else if (op == SUB)
	// 	k_substraction_element_wise<<<blocks, NB_THREADS>>>(d_vec, len, n);
	// else if (op == MUL)
	// 	k_multiplication_element_wise<<<blocks, NB_THREADS>>>(d_vec, len, n);
	// else if (op == DIV)
	// 	k_division_element_wise<<<blocks, NB_THREADS>>>(d_vec, len, n);
	// else if (op == MOD)
	// 	k_modulo_element_wise<<<blocks, NB_THREADS>>>(d_vec, len, n);

	hipDeviceSynchronize();// checkCudaErrors(cudaGetLastError());
	checkCudaErrors(hipMemcpy(h_result, d_result, sizeof(int) * len, hipMemcpyDeviceToHost));
	hipFree(d_vec1_);
	hipFree(d_vec2_);
	hipFree(d_result_);
	return (h_result);
}
