#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define NB_THREADS 50

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

enum ft_op {ADD = 0, SUB, MUL, DIV, MOD};



template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}


__global__ void	k_addition_element_wise(int *values, int len, int n) {

	int			id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < len)
		values[id] += n;
}

__global__ void	k_substraction_element_wise(int *values, int len, int n) {

	int			id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < len)
		values[id] -= n;
}

__global__ void	k_multiplication_element_wise(int *values, int len, int n) {

	int			id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < len)
		values[id] *= n;
}

__global__ void	k_division_element_wise(int *values, int len, int n) {

	int			id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < len)
		values[id] /= n;
}

__global__ void	k_modulo_element_wise(int *values, int len, int n) {

	int			id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id < len)
		values[id] = values[id] % n;
}

void			element_wise(int *v, int len, int n, ft_op op) {

	int		*d_values;
	int		**d_values_ = &d_values;

	int		blocks = (len % NB_THREADS == 0) ? (len / NB_THREADS):(len / NB_THREADS) + 1;

	checkCudaErrors(hipMalloc(d_values_, sizeof(int) * len));
	checkCudaErrors(hipMemcpy(d_values, v, sizeof(int) * len, hipMemcpyHostToDevice));

	if (op == ADD)
		k_addition_element_wise<<<blocks, NB_THREADS>>>(d_values, len, n);
	else if (op == SUB)
		k_substraction_element_wise<<<blocks, NB_THREADS>>>(d_values, len, n);
	else if (op == MUL)
		k_multiplication_element_wise<<<blocks, NB_THREADS>>>(d_values, len, n);
	else if (op == DIV)
		k_division_element_wise<<<blocks, NB_THREADS>>>(d_values, len, n);
	else if (op == MOD)
		k_modulo_element_wise<<<blocks, NB_THREADS>>>(d_values, len, n);

	hipDeviceSynchronize();// checkCudaErrors(cudaGetLastError());
	checkCudaErrors(hipMemcpy(v, d_values, sizeof(int) * len, hipMemcpyDeviceToHost));
	hipFree(d_values_);
}
